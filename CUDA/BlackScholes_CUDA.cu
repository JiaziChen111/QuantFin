/**
* Copyright 2014 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/**Modified, updated and re-oragnized some part of codes 
* by Dr. Yingfeng Yu, CUEB, School of Finance,
* last modified Jul,4th,2016
* Only for education purpose
*/

#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"

const int OPT_N = 6400000;

const int  NUM_ITERATIONS = 10;

const int          OPT_SZ = OPT_N * sizeof(float);
const float      RISKFREE = 0.02f;
const float    VOLATILITY = 0.30f;

#define DIV_UP(a, b) ( ((a) + (b) - 1) / (b) )

float RandFloat(float low, float high)
{
    float t = (float)rand() / (float)RAND_MAX;
    return (1.0f - t) * low + t * high;
}

__device__ inline float cndGPUv1(float d)
{//written by Dr.Yingfeng Yu
    
    float cnd;
    cnd=normcdff(d);
    return cnd;
}

__device__ inline float cndGPUv2(float d)
{//Nvidia's code
    const float       A1 = 0.31938153f;
    const float       A2 = -0.356563782f;
    const float       A3 = 1.781477937f;
    const float       A4 = -1.821255978f;
    const float       A5 = 1.330274429f;
    const float RSQRT2PI = 0.39894228040143267793994605993438f;

    float K, cnd;

    K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

    cnd = RSQRT2PI * __expf(- 0.5f * d * d) * (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0f - cnd;
    return cnd;
}

__device__ inline float pndGPU(float d)
{//written by Dr.Yingfeng Yu
    const float PI = 3.141592653589793238462643f;
    float pnd;
    pnd= (rsqrtf(2.0f*PI))*__expf(-0.5f*d*d);
    return pnd;
}


__device__ inline void BlackScholesBodyGPU(
    float &CallResult,
    float &PutResult,
    float &DeltaCall, //new added by yyf
    float &Gamma,//new added by yyf 
    float S, //Stock price
    float X, //Option strike
    float T, //Option years
    float R, //Riskless rate
    float V  //Volatility rate
)
{
    float sqrtT, expRT;
    float d1, d2,CNDD1, CNDD2;

    sqrtT = __fdividef(1.0F, rsqrtf(T));
    d1 = __fdividef(__logf(S / X) + (R + 0.5f * V * V) * T, V * sqrtT);
    d2 = d1 - V * sqrtT;

    CNDD1 = cndGPUv1(d1);// it is better to use my version
    CNDD2 = cndGPUv1(d2);

    //Calculate Call and Put simultaneously
    expRT = __expf(- R * T);
    CallResult = S * CNDD1 - X * expRT * CNDD2;
    PutResult  = X * expRT * (1.0f - CNDD2) - S * (1.0f - CNDD1);
    DeltaCall = pndGPU(d1);
    Gamma = __fdividef(pndGPU(d1),V*S*sqrtT);
}


////////////////////////////////////////////////////////////////////////////////
//Process an array of optN options on GPU
////////////////////////////////////////////////////////////////////////////////
__launch_bounds__(128)
__global__ void BlackScholesGPU(
    float * d_CallResult,
    float * d_PutResult,
    float * d_DeltaCall,
    float * d_Gamma,
    float * d_StockPrice,
    float * d_OptionStrike,
    float * d_OptionYears,
    float Riskfree,
    float Volatility,
    int optN
)
{
    const int opt = blockDim.x * blockIdx.x + threadIdx.x;
    float callResult, putResult, deltaCall, gamma;//add yyf

    BlackScholesBodyGPU(
            callResult,
            putResult,
            deltaCall,
            gamma,
            d_StockPrice[opt],
            d_OptionStrike[opt],
            d_OptionYears[opt],
            Riskfree,
            Volatility);
        d_CallResult[opt] = callResult;
        d_PutResult[opt] = putResult;
        d_DeltaCall[opt] = deltaCall;
        d_Gamma[opt] = gamma;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    printf("[%s] - Starting...\n", argv[0]);

    float
        *h_CallResult,
        *h_PutResult,
        *h_DeltaCall,
        *h_Gamma,
        //CPU instance of input data
        *h_StockPrice,
        *h_OptionStrike,
        *h_OptionYears;

    float
    //Results calculated by GPU
        *d_CallResult,
        *d_PutResult,
        *d_DeltaCall,
        *d_Gamma,
    //GPU instance of input data
        *d_StockPrice,
        *d_OptionStrike,
        *d_OptionYears;

    int i;

    printf("Initializing data...\n");

    h_CallResult   =   (float *)malloc(OPT_SZ);
    h_PutResult   =   (float *)malloc(OPT_SZ);
    h_DeltaCall     =   (float *)malloc(OPT_SZ);
    h_Gamma       =   (float *)malloc(OPT_SZ);
    h_StockPrice           =   (float *)malloc(OPT_SZ);
    h_OptionStrike      =   (float *)malloc(OPT_SZ);
    h_OptionYears       =   (float *)malloc(OPT_SZ);

    printf("...allocating GPU memory for options.\n");
    hipMalloc( (void **)    &d_CallResult,          OPT_SZ);
    hipMalloc( (void **)    &d_PutResult,          OPT_SZ);
    hipMalloc( (void **)    &d_DeltaCall,            OPT_SZ);
    hipMalloc( (void **)    &d_Gamma,              OPT_SZ);
    hipMalloc( (void **)    &d_StockPrice,         OPT_SZ);
    hipMalloc( (void **)    &d_OptionStrike,    OPT_SZ);
    hipMalloc( (void **)    &d_OptionYears,     OPT_SZ);

    printf("...generating input data in CPU mem.\n");
    srand(5347);

    //Generate options set
    for (i = 0; i < OPT_N; i++)
    {
        h_StockPrice[i]     =       RandFloat(5.0f, 30.0f);
        h_OptionStrike[i]  =      RandFloat(1.0f, 100.0f);
        h_OptionYears[i]   =      RandFloat(0.25f, 10.0f);
    }

    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    hipMemcpy(d_StockPrice,  h_StockPrice,   OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionStrike, h_OptionStrike,  OPT_SZ, hipMemcpyHostToDevice);
    hipMemcpy(d_OptionYears,  h_OptionYears,   OPT_SZ, hipMemcpyHostToDevice);
    printf("Data init done.\n\n");


    printf("Executing Black-Scholes GPU kernel (%i iterations)...\n", NUM_ITERATIONS);
    hipDeviceSynchronize();

    for (i = 0; i < NUM_ITERATIONS; i++)
    {
            printf("Now executing Black-Scholes GPU kernel (%i -th)...\n", i);
        BlackScholesGPU<<<DIV_UP(OPT_N, 128), 128/*480, 128*/>>>(
            d_CallResult,
            d_PutResult,
            d_DeltaCall, //new added by yyf
            d_Gamma,//new added  by yyf
            d_StockPrice,
            d_OptionStrike,
            d_OptionYears,
            RISKFREE,
            VOLATILITY,
            OPT_N
        );
    }

    hipDeviceSynchronize();
    hipMemcpy(h_CallResult, d_CallResult, OPT_SZ, hipMemcpyDeviceToHost);
    hipMemcpy(h_PutResult,  d_PutResult,  OPT_SZ, hipMemcpyDeviceToHost);
    hipMemcpy(h_DeltaCall,  d_DeltaCall,  OPT_SZ, hipMemcpyDeviceToHost);
     hipMemcpy(h_Gamma,  d_Gamma,  OPT_SZ, hipMemcpyDeviceToHost);

    int NN=20;
    printf("===============================================Basic Info===============================================\n");
    printf("\t\t\tTotal Num.of Options=%d,\n\t\t\tRisk-free rate Rf=%f,\n\t\t\tVolatility(Sigma)=%f.\n",OPT_N,RISKFREE,VOLATILITY);
    printf("\t\t\tCopyright belongs to Nvidia, modified by Dr. Yingfeng Yu. \n\t\t\tFor education purpose only.\n" );
    printf("=====================CUDA Results========================================||=====BSM's other Info========\n");
    printf("The index    |\tCall Price    |\t Put Price    |\tCall Delta|\tGamma\t ||         (S,K,T)\n");

    for (i = 0; i < NN; i++)
    {
       printf("[%d]\t\t %f\t %f\t %f\t %f||(%f,%f,%f)\n",i+1,
            h_CallResult[i],h_PutResult[i],
            h_DeltaCall[i],h_Gamma[i],
            h_StockPrice[i],h_OptionStrike[i],h_OptionYears[i]);
    }
    printf(".\n");
    printf(".\n");
    printf(".\n");

    for (i = OPT_N-NN; i < OPT_N; i++)
    {
        printf("[%d]\t %f\t %f\t %f\t %f||(%f,%f,%f)\n",i+1,
            h_CallResult[i],h_PutResult[i],
            h_DeltaCall[i],h_Gamma[i],
            h_StockPrice[i],h_OptionStrike[i],h_OptionYears[i]);
    }
    
    printf("...releasing GPU memory.\n");
    hipFree(d_OptionYears);
    hipFree(d_OptionStrike);
    hipFree(d_StockPrice);
    hipFree(d_PutResult);
    hipFree(d_CallResult);
    hipFree(d_DeltaCall);
    hipFree(d_Gamma);

    printf("...releasing CPU memory.\n");
    free(h_OptionYears);
    free(h_OptionStrike);
    free(h_StockPrice);
    free(h_PutResult);
    free(h_CallResult);
    free(h_DeltaCall);
    free(h_Gamma);

    printf("All testing, .... done.\n");
    hipDeviceReset();
}
